#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void a_array_b(int * d_out, int * d_in, int size)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if(myId<size){
    int p = d_in[myId]/100;
    atomicAdd(&(d_out[p]), 1);}
    __syncthreads(); 
}
__global__ void b_array_b(int * b_d_out, int * b_d_in, int size)
{   
    extern __shared__ int sdata[];
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if(myId<size){
    int p = b_d_in[myId]/100;
    atomicAdd(&(sdata[p]), 1);}
    __syncthreads(); 
   for(int i =0;i<10;i++){
	b_d_out[i+bid*10]=sdata[i];}
}
__global__ void b_add_b(int * b_d_out, int * b_d_in, int index)
{   
    int tid = threadIdx.x;
    for(int i=0; i< index;i++){
     b_d_out[tid] += b_d_in[i*10+tid];}
    __syncthreads();  
}
__global__ void array_c(int * d_out, int * d_in, int size)
{   
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
	int val;
	d_out[myId] = d_in[myId];
	__syncthreads(); 
	for(int d = 1;d<size;d=d*2){ 
        if(myId>=d){
		 val = d_out[myId-d];}	
        __syncthreads();
		if(myId>=d)
		  d_out[myId] += val;
		__syncthreads();
		}
}


int main(void)
{   
    int v[1000000];   
    FILE *infile;
    infile = fopen("inp.txt","r");
    if(infile==NULL){
        printf("error");
        exit(1);
    }
        char c;
	int size = 0;
    while(fscanf(infile,"%d,%c",&v[size],&c)!=EOF)
	size++;
    fclose(infile);
    int h_in[size]; 
    for(int i=0; i<size; i++){
	h_in[i]=v[i];
    }
    const int ARRAY_SIZE = size;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);	
    const int maxThreadsPerBlock = 500;
    int threads = maxThreadsPerBlock;
    int blocks;
    if(ARRAY_SIZE % maxThreadsPerBlock==0) blocks = ARRAY_SIZE / maxThreadsPerBlock;
    else blocks = ARRAY_SIZE / maxThreadsPerBlock+1;

    // declare GPU memory pointers
    int * d_in, *d_out, *c_out, * b_d_out, * b_d_intermediate;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, 10*sizeof(int)); 
    hipMalloc((void **) &c_out, 10*sizeof(int)); 
    hipMalloc((void **) &b_d_out, 10*sizeof(int)); 
    hipMalloc((void **) &b_d_intermediate, ARRAY_BYTES); // overallocated

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    a_array_b<<<blocks, threads>>>(d_out, d_in,ARRAY_SIZE);
    array_c<<<blocks, threads>>>(c_out,d_out,10);
    b_array_b<<<blocks, threads, 10*sizeof(int)>>>(b_d_intermediate, d_in, ARRAY_SIZE);
    int index = blocks;
    threads = 10;
    blocks=1;
    b_add_b<<<blocks,10>>>(b_d_out, b_d_intermediate, index);


    // copy back the sum from GPU
    int b[10];
    hipMemcpy(b, d_out, 10*sizeof(int), hipMemcpyDeviceToHost);
	FILE *fpa = fopen("q2a.txt","w");
	
    for(int i=0; i<10; i++){
    fprintf(fpa,"%d,",b[i]);}
	fclose(fpa);
	
    int cc[10];
    hipMemcpy(cc, c_out, 10*sizeof(int), hipMemcpyDeviceToHost);
    FILE *fpc = fopen("q2c.txt","w");
	
    for(int i=0; i<10; i++){
    fprintf(fpc,"%d,",cc[i]);}
	fclose(fpc);

    hipFree(d_out);
	hipFree(c_out);
	
     // copy back the sum from GPU
    int bb[10];
    hipMemcpy(bb, b_d_out, 10*sizeof(int), hipMemcpyDeviceToHost);
	FILE *fpb = fopen("q2b.txt","w");
    for(int i=0; i<10; i++){
     fprintf(fpb, "%d,",bb[i]);}
	 fclose(fpb);
	 
    hipFree(d_in);
    hipFree(b_d_out); 
     hipFree(b_d_intermediate);
    return 0;
}
